#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * Vector multiplication: C = A * B.
 *
 * This sample is a very basic sample that implements element by element
 * vector multiplication. It is based on the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include "../not_includes/complex.h"

/*
 * CUDA Kernel Device code
 *
 * Computes the vector product of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorMult(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] * B[i];
    }
}

float deviceMultiply(float a, float b) { return a * b; }

std::tuple<float *, float *, float *> allocateHostMemory(int numElements) {
    size_t size = numElements * sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
    }

    return {h_A, h_B, h_C};
}

std::tuple<float *, float *, float *> allocateDeviceMemory(int numElements) {
    // Allocate the device input vector A
    float *d_A = NULL;
    size_t size = numElements * sizeof(float);
    hipError_t err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return {d_A, d_B, d_C};
}

void copyFromHostToDevice(float *h_A, float *h_B, float *d_A, float *d_B, int numElements) {
    size_t size = numElements * sizeof(float);
    // Copy the host input vectors A and B in host memory to the device input vectors in device
    // memory
    printf("Copy input data from the host memory to the CUDA device\n");
    hipError_t err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

void executeKernel(float *d_A, float *d_B, float *d_C, int numElements) {
    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // REPLACE x, y, z with a, b, and c variables for memory on the GPU
    vectorMult<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipError_t err = hipGetLastError();

    if (err != hipSuccess) {
        // fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
        // hipGetErrorString(err)); exit(EXIT_FAILURE);
    }
}

__host__ void copyFromDeviceToHost(float *d_C, float *h_C, int numElements) {
    size_t size = numElements * sizeof(float);
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    hipError_t err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n",
        hipGetErrorString(err)); exit(EXIT_FAILURE);
    }
}

// Free device global memory
void deallocateMemory(float *h_A, float *h_B, float *h_C, float *d_A, float *d_B, float *d_C) {
    // Error code to check return values for CUDA calls
    hipError_t err = hipFree(d_A);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
        hipGetErrorString(err)); exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
        hipGetErrorString(err)); exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
        hipGetErrorString(err)); exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
}

// Reset the device and exit
void cleanUpDevice() {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipError_t err = hipDeviceReset();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
        hipGetErrorString(err)); exit(EXIT_FAILURE);
    }
}

void performTest(float *h_A, float *h_B, float *h_C, int numElements) {
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs((h_A[i] * h_B[i]) - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");
}

/*
 * Host main routine
 */
int main(void) {
    int numElements = 50000;
    printf("[Vector multiplication of %d elements]\n", numElements);

    auto [h_A, h_B, h_C] = allocateHostMemory(numElements);
    auto [d_A, d_B, d_C] = allocateDeviceMemory(numElements);
    copyFromHostToDevice(h_A, h_B, d_A, d_B, numElements);

    executeKernel(d_A, d_B, d_C, numElements);

    copyFromDeviceToHost(d_C, h_C, numElements);
    performTest(h_A, h_B, h_C, numElements);
    deallocateMemory(h_A, h_B, h_C, d_A, d_B, d_C);

    cleanUpDevice();
    printf("Done\n");
    return 0;
}


