
#include <hip/hip_runtime.h>
// Vector addition (device code)

#define N 100

extern "C" __global__ void matSum(int *a, int *b, int *c)
{
    int tid = blockIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}